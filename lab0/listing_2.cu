#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include ""
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
		}\
}

__global__ void PairSwap(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	char temp;
	if (idx % 2 == 0)
		if (idx < fsize && input_gpu[idx] != '\0') {
			if ((input_gpu[idx] >= 65 && input_gpu[idx] <= 90) ||
				(input_gpu[idx] >= 97 && input_gpu[idx] <= 122))
				if ((input_gpu[idx + 1] >= 65 && input_gpu[idx + 1] <= 90) ||
					(input_gpu[idx + 1] >= 97 && input_gpu[idx + 1] <= 122)){
					temp = input_gpu[idx + 1];
					input_gpu[idx + 1] = input_gpu[idx];
					input_gpu[idx] = temp;
				}
		}
}

int main(int argc, char **argv) {
	// init, and check
	/*if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}*/
	//FILE *fp = fopen(argc[1], "r");
	FILE *fp = fopen("test.txt", "r");
	if (fp == NULL) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);
	// read files
	MemoryBuffer<char> text(fsize + 1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);               
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	int blocksize = 8;
	int nblock = (fsize / blocksize) + (fsize % blocksize == 0 ? 0 : 1);
	//pair swap
	PairSwap << < nblock, blocksize >> >(input_gpu, fsize);
	puts(text_smem.get_cpu_ro());

	return 0;
}